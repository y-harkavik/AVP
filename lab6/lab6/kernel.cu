#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "helper_image.h"

#include <iostream> 
#include <hip/hip_runtime.h> 
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <intrin.h>
#include <windows.h>
#include <ctime>
#include <cmath>
#include <cstdlib>
#include <tuple>

#pragma comment(lib, "cudart") 

#define BLOCK_SIZE_X 1024
#define BLOCK_SIZE_Y 1
//#define IMAGE_WIDTH 11880
//#define IMAGE_HEIGHT 8648
#define IMAGE_WIDTH 6000
#define IMAGE_HEIGHT 4000
#define MAX_BLOCKS 50000

using namespace std;

void cpu_filterImage(BYTE*, BYTE*, int, int);
void cuda_filterImage(BYTE*, BYTE*, bool);
void cudaCheckStatus(hipError_t);
void resizeImage(BYTE*, BYTE*, int, int);
bool checkEquality(BYTE*, BYTE*, int, int);
__global__ void cuda_filterImage(BYTE*, BYTE*, size_t, size_t);
__global__ void cuda_filterImageShared(BYTE*, BYTE*, size_t, size_t);
__device__ BYTE sumPixels(BYTE, BYTE, BYTE, BYTE, BYTE, BYTE, BYTE, BYTE, BYTE);
__device__ WORD pack(uchar3);
__device__ uchar2 unpack(WORD);

int main() {
	unsigned int imageWidth = 0, imageHeight = 0, channels = 0;

	const char primaryImagePath[] = "D:\\image.ppm";
	const char outputImageCpuPath[] = "D:\\imageCPU.ppm";
	const char outputImageGpuPath[] = "D:\\imageGPU.ppm";
	const char outputImageGpuSharedPath[] = "D:\\imageGPUshared.ppm";

	BYTE *primaryImage = NULL;

	__loadPPM(primaryImagePath, &primaryImage, &imageWidth, &imageHeight, &channels);

	auto* outputImageCpu = (BYTE*)malloc(imageWidth * imageHeight * sizeof(BYTE) * 3);
	auto* outputImageGpu = (BYTE*)malloc(imageWidth * imageHeight * sizeof(BYTE) * 3);
	auto* outputImageGpuShared = (BYTE*)malloc(imageWidth * imageHeight * sizeof(BYTE) * 3);
	auto* resizedImage = (BYTE*)malloc((imageWidth + 2) * (imageHeight + 2) * sizeof(BYTE) * 3);

	resizeImage(primaryImage, resizedImage, imageWidth, imageHeight);

	cpu_filterImage(resizedImage, outputImageCpu, imageWidth, imageHeight);
	cuda_filterImage(resizedImage, outputImageGpu, false);
	cuda_filterImage(resizedImage, outputImageGpuShared, true);

	__savePPM(outputImageCpuPath, outputImageCpu, imageWidth, imageHeight, channels);
	__savePPM(outputImageGpuPath, outputImageGpu, imageWidth, imageHeight, channels);
	__savePPM(outputImageGpuSharedPath, outputImageGpuShared, imageWidth, imageHeight, channels);

	cout << "Start compare" << endl;

	if (checkEquality(outputImageCpu, outputImageGpu, IMAGE_WIDTH, IMAGE_HEIGHT)
		&& checkEquality(outputImageGpu, outputImageGpuShared, IMAGE_WIDTH, IMAGE_HEIGHT)) {
		cout << "Results are equals!" << endl;
	}
	else {
		cout << "Results are NOT equals!" << endl;
	}

	free(primaryImage);
	free(resizedImage);
	free(outputImageCpu);
	free(outputImageGpu);
	free(outputImageGpuShared);
}

void cuda_filterImage(BYTE* inMatrix, BYTE* outMatrix, bool optimizationFlag) {
	float resultTime;

	BYTE* device_inMatrix;
	BYTE* device_outMatrix;

	hipEvent_t cuda_startTime;
	hipEvent_t cuda_endTime;

	cudaCheckStatus(hipEventCreate(&cuda_startTime));
	cudaCheckStatus(hipEventCreate(&cuda_endTime));

	int numOfBlocksInRow = (int)ceil((double)(IMAGE_WIDTH) / (BLOCK_SIZE_X * 2));
	int numOfBlockInColumn = IMAGE_HEIGHT;
	int blocksNeeded = numOfBlockInColumn * numOfBlocksInRow;
	int maxBlocksPerIteration = MAX_BLOCKS - MAX_BLOCKS % numOfBlocksInRow;

	for (int i = 0, int times = 1; i < blocksNeeded; i += maxBlocksPerIteration, times++) {
		int blocksInIteration = (blocksNeeded - i) < maxBlocksPerIteration ? blocksNeeded - i : maxBlocksPerIteration;
		size_t pitchInMatrix = 0, pitchOutMatrix = 0;
		int gridSizeY = blocksInIteration / numOfBlocksInRow;
		int gridSizeX = numOfBlocksInRow;

		cudaCheckStatus(hipMallocPitch((void**)&device_inMatrix, &pitchInMatrix, (IMAGE_WIDTH + 2) * 3, gridSizeY + 2));
		cudaCheckStatus(hipMallocPitch((void**)&device_outMatrix, &pitchOutMatrix, IMAGE_WIDTH * 3, gridSizeY));
		cudaCheckStatus(hipMemcpy2D(
			device_inMatrix, pitchInMatrix,
			inMatrix, (IMAGE_WIDTH + 2) * 3,
			(IMAGE_WIDTH + 2) * 3, gridSizeY + 2,
			hipMemcpyHostToDevice));

		dim3 blockSize(BLOCK_SIZE_X);
		dim3 gridSize(gridSizeX, gridSizeY);

		cudaCheckStatus(hipEventRecord(cuda_startTime, NULL));

		if (optimizationFlag) {
			cuda_filterImageShared << < gridSize, blockSize >> > (device_inMatrix, device_outMatrix, pitchInMatrix, pitchOutMatrix);
		}
		else {
			cuda_filterImage << < gridSize, blockSize >> > (device_inMatrix, device_outMatrix, pitchInMatrix, pitchOutMatrix);
		}

		cudaCheckStatus(hipPeekAtLastError());
		cudaCheckStatus(hipEventRecord(cuda_endTime, NULL));
		cudaCheckStatus(hipEventSynchronize(cuda_endTime));

		cudaCheckStatus(hipEventElapsedTime(&resultTime, cuda_startTime, cuda_endTime));

		if (optimizationFlag) {
			printf("%d: CUDA time with optimization: %lf seconds\n", times, (double)resultTime / CLOCKS_PER_SEC);
		}
		else {
			printf("%d: CUDA time: %lf seconds\n", times, (double)resultTime / CLOCKS_PER_SEC);
		}

		cudaCheckStatus(hipMemcpy2D(
			outMatrix, IMAGE_WIDTH * 3,
			device_outMatrix, pitchOutMatrix,
			IMAGE_WIDTH * 3, gridSizeY,
			hipMemcpyDeviceToHost)
		);

		inMatrix = &inMatrix[(IMAGE_WIDTH + 2) * gridSizeY * times * 3];
		outMatrix = &outMatrix[IMAGE_WIDTH * gridSizeY * times * 3];

		cudaCheckStatus(hipFree(device_inMatrix));
		cudaCheckStatus(hipFree(device_outMatrix));
	}
}

void cpu_filterImage(BYTE* primaryImage, BYTE* outputImage, int imageWidth, int imageHeight) {
	primaryImage = &primaryImage[(imageWidth + 2 + 1) * 3];

	clock_t startTime, endTime;
	startTime = clock();
	for (auto i = 0; i < imageHeight; i++) {
		for (auto j = 0; j < imageWidth; j++) {
			for (auto k = 0; k < 3; k++) {
				short sum = 0;
				int index = 0;

				index = (i * (imageWidth + 2) + j) * 3 + k;
				sum += primaryImage[(i * (imageWidth + 2) + j) * 3 + k];
				sum += primaryImage[(i * (imageWidth + 2) + j + 1) * 3 + k];
				sum += primaryImage[(i * (imageWidth + 2) + j - 1) * 3 + k];

				sum += primaryImage[((i + 1) * (imageWidth + 2) + j) * 3 + k];
				sum += primaryImage[((i + 1) * (imageWidth + 2) + j + 1) * 3 + k];
				sum += primaryImage[((i + 1) * (imageWidth + 2) + j - 1) * 3 + k];

				sum += primaryImage[((i - 1) * (imageWidth + 2) + j) * 3 + k];
				sum += primaryImage[((i - 1) * (imageWidth + 2) + j + 1) * 3 + k];
				sum += primaryImage[((i - 1) * (imageWidth + 2) + j - 1) * 3 + k];

				sum = sum / 9;

				if (sum > 255) {
					sum = 255;
				}

				outputImage[(i * imageWidth + j) * 3 + k] = (BYTE)sum;
			}
		}
	}
	endTime = clock();
	printf("-  CPU time: %lf seconds\n", (double)(endTime - startTime) / CLOCKS_PER_SEC);
}

__global__ void cuda_filterImage(BYTE* inMatrix, BYTE* outMatrix, size_t pitchInMatrix, size_t pitchOutMatrix) {
	int remainderElements = (IMAGE_WIDTH % (blockDim.x * 2)) / 2;

	if (remainderElements != 0 && (blockIdx.x + 1) % gridDim.x == 0 && threadIdx.x >= remainderElements) {
		return;
	}

	WORD *startOfProcessingRow = (WORD*)&inMatrix[pitchInMatrix * blockIdx.y + blockIdx.x * blockDim.x * 2 * 3 + threadIdx.x * 2 * 3];

	WORD a1 = startOfProcessingRow[0];
	WORD a2 = startOfProcessingRow[1];
	WORD a3 = startOfProcessingRow[2];
	WORD a4 = startOfProcessingRow[3];
	WORD a5 = startOfProcessingRow[4];
	WORD a6 = startOfProcessingRow[5];

	WORD b1 = startOfProcessingRow[pitchInMatrix / 2];
	WORD b2 = startOfProcessingRow[pitchInMatrix / 2 + 1];
	WORD b3 = startOfProcessingRow[pitchInMatrix / 2 + 2];
	WORD b4 = startOfProcessingRow[pitchInMatrix / 2 + 3];
	WORD b5 = startOfProcessingRow[pitchInMatrix / 2 + 4];
	WORD b6 = startOfProcessingRow[pitchInMatrix / 2 + 5];

	WORD c1 = startOfProcessingRow[pitchInMatrix];
	WORD c2 = startOfProcessingRow[pitchInMatrix + 1];
	WORD c3 = startOfProcessingRow[pitchInMatrix + 2];
	WORD c4 = startOfProcessingRow[pitchInMatrix + 3];
	WORD c5 = startOfProcessingRow[pitchInMatrix + 4];
	WORD c6 = startOfProcessingRow[pitchInMatrix + 5];

	uchar2 aa1 = unpack(a1);
	uchar2 aa2 = unpack(a2);
	uchar2 aa3 = unpack(a3);
	uchar2 aa4 = unpack(a4);
	uchar2 aa5 = unpack(a5);
	uchar2 aa6 = unpack(a6);

	uchar2 bb1 = unpack(b1);
	uchar2 bb2 = unpack(b2);
	uchar2 bb3 = unpack(b3);
	uchar2 bb4 = unpack(b4);
	uchar2 bb5 = unpack(b5);
	uchar2 bb6 = unpack(b6);

	uchar2 cc1 = unpack(c1);
	uchar2 cc2 = unpack(c2);
	uchar2 cc3 = unpack(c3);
	uchar2 cc4 = unpack(c4);
	uchar2 cc5 = unpack(c5);
	uchar2 cc6 = unpack(c6);


	uchar3 firstPixel, secondPixel;
	firstPixel.x = sumPixels(aa1.x, aa2.y, aa4.x, bb1.x, bb2.y, bb4.x, cc1.x, cc2.y, cc4.x);
	firstPixel.y = sumPixels(aa1.y, aa3.x, aa4.y, bb1.y, bb3.x, bb4.y, cc1.y, cc3.x, cc4.y);
	firstPixel.z = sumPixels(aa2.x, aa3.y, aa5.x, bb2.x, bb3.y, bb5.x, cc2.x, cc3.y, cc5.x);
	secondPixel.x = sumPixels(aa2.y, aa4.x, aa5.y, bb2.y, bb4.x, bb5.y, cc2.y, cc4.x, cc5.y);
	secondPixel.y = sumPixels(aa3.x, aa4.y, aa6.x, bb3.x, bb4.y, bb6.x, cc3.x, cc4.y, cc6.x);
	secondPixel.z = sumPixels(aa3.y, aa5.x, aa6.y, bb3.y, bb5.x, bb6.y, cc3.y, cc5.x, cc6.y);

	outMatrix = &outMatrix[blockIdx.y * pitchOutMatrix + threadIdx.x * 2 * 3 + blockIdx.x * blockDim.x * 2 * 3];
	outMatrix[0] = firstPixel.x;
	outMatrix[1] = firstPixel.y;
	outMatrix[2] = firstPixel.z;
	outMatrix[3] = secondPixel.x;
	outMatrix[4] = secondPixel.y;
	outMatrix[5] = secondPixel.z;
}

__global__ void cuda_filterImageShared(BYTE* inMatrix, BYTE* outMatrix, size_t pitchInMatrix, size_t pitchOutMatrix) {
	int remainderElements = (IMAGE_WIDTH % (blockDim.x * 2)) / 2;

	if (remainderElements != 0 && (blockIdx.x + 1) % gridDim.x == 0 && threadIdx.x >= remainderElements) {
		return;
	}

	__shared__ WORD sharedMemoryIn[3][(BLOCK_SIZE_X + 1) * 3];
	__shared__ WORD sharedMemoryOut[BLOCK_SIZE_X * 3];

	WORD *startOfProcessingRow = (WORD*)&inMatrix[blockIdx.y * pitchInMatrix + blockIdx.x * blockDim.x * 2 * 3];
	WORD *outputRow = (WORD*)&outMatrix[blockIdx.y * pitchOutMatrix + blockIdx.x * blockDim.x * 2 * 3];

	if (threadIdx.x == 0) {
		WORD *tempPointer = &startOfProcessingRow[threadIdx.x];

		sharedMemoryIn[0][threadIdx.x] = tempPointer[0];
		sharedMemoryIn[0][threadIdx.x + 1] = tempPointer[1];
		sharedMemoryIn[0][threadIdx.x + 2] = tempPointer[2];

		sharedMemoryIn[1][threadIdx.x] = tempPointer[pitchInMatrix / 2];
		sharedMemoryIn[1][threadIdx.x + 1] = tempPointer[pitchInMatrix / 2 + 1];
		sharedMemoryIn[1][threadIdx.x + 2] = tempPointer[pitchInMatrix / 2 + 2];

		sharedMemoryIn[2][threadIdx.x] = tempPointer[pitchInMatrix];
		sharedMemoryIn[2][threadIdx.x + 1] = tempPointer[pitchInMatrix + 1];
		sharedMemoryIn[2][threadIdx.x + 2] = tempPointer[pitchInMatrix + 2];
	}

	startOfProcessingRow = &startOfProcessingRow[(threadIdx.x + 1) * 3];

	sharedMemoryIn[0][threadIdx.x * 3 + 3] = startOfProcessingRow[0];
	sharedMemoryIn[0][threadIdx.x * 3 + 4] = startOfProcessingRow[1];
	sharedMemoryIn[0][threadIdx.x * 3 + 5] = startOfProcessingRow[2];

	sharedMemoryIn[1][threadIdx.x * 3 + 3] = startOfProcessingRow[pitchInMatrix / 2];
	sharedMemoryIn[1][threadIdx.x * 3 + 4] = startOfProcessingRow[pitchInMatrix / 2 + 1];
	sharedMemoryIn[1][threadIdx.x * 3 + 5] = startOfProcessingRow[pitchInMatrix / 2 + 2];

	sharedMemoryIn[2][threadIdx.x * 3 + 3] = startOfProcessingRow[pitchInMatrix];
	sharedMemoryIn[2][threadIdx.x * 3 + 4] = startOfProcessingRow[pitchInMatrix + 1];
	sharedMemoryIn[2][threadIdx.x * 3 + 5] = startOfProcessingRow[pitchInMatrix + 2];


	__syncthreads();

	WORD a1 = sharedMemoryIn[0][threadIdx.x * 3];
	WORD a2 = sharedMemoryIn[0][threadIdx.x * 3 + 1];
	WORD a3 = sharedMemoryIn[0][threadIdx.x * 3 + 2];
	WORD a4 = sharedMemoryIn[0][threadIdx.x * 3 + 3];
	WORD a5 = sharedMemoryIn[0][threadIdx.x * 3 + 4];
	WORD a6 = sharedMemoryIn[0][threadIdx.x * 3 + 5];

	WORD b1 = sharedMemoryIn[1][threadIdx.x * 3];
	WORD b2 = sharedMemoryIn[1][threadIdx.x * 3 + 1];
	WORD b3 = sharedMemoryIn[1][threadIdx.x * 3 + 2];
	WORD b4 = sharedMemoryIn[1][threadIdx.x * 3 + 3];
	WORD b5 = sharedMemoryIn[1][threadIdx.x * 3 + 4];
	WORD b6 = sharedMemoryIn[1][threadIdx.x * 3 + 5];

	WORD c1 = sharedMemoryIn[2][threadIdx.x * 3];
	WORD c2 = sharedMemoryIn[2][threadIdx.x * 3 + 1];
	WORD c3 = sharedMemoryIn[2][threadIdx.x * 3 + 2];
	WORD c4 = sharedMemoryIn[2][threadIdx.x * 3 + 3];
	WORD c5 = sharedMemoryIn[2][threadIdx.x * 3 + 4];
	WORD c6 = sharedMemoryIn[2][threadIdx.x * 3 + 5];

	uchar2 aa1 = unpack(a1);
	uchar2 aa2 = unpack(a2);
	uchar2 aa3 = unpack(a3);
	uchar2 aa4 = unpack(a4);
	uchar2 aa5 = unpack(a5);
	uchar2 aa6 = unpack(a6);

	uchar2 bb1 = unpack(b1);
	uchar2 bb2 = unpack(b2);
	uchar2 bb3 = unpack(b3);
	uchar2 bb4 = unpack(b4);
	uchar2 bb5 = unpack(b5);
	uchar2 bb6 = unpack(b6);

	uchar2 cc1 = unpack(c1);
	uchar2 cc2 = unpack(c2);
	uchar2 cc3 = unpack(c3);
	uchar2 cc4 = unpack(c4);
	uchar2 cc5 = unpack(c5);
	uchar2 cc6 = unpack(c6);

	uchar3 firstPixel, secondPixel;
	firstPixel.x = sumPixels(aa1.x, aa2.y, aa4.x, bb1.x, bb2.y, bb4.x, cc1.x, cc2.y, cc4.x);
	firstPixel.y = sumPixels(aa1.y, aa3.x, aa4.y, bb1.y, bb3.x, bb4.y, cc1.y, cc3.x, cc4.y);
	firstPixel.z = sumPixels(aa2.x, aa3.y, aa5.x, bb2.x, bb3.y, bb5.x, cc2.x, cc3.y, cc5.x);
	secondPixel.x = sumPixels(aa2.y, aa4.x, aa5.y, bb2.y, bb4.x, bb5.y, cc2.y, cc4.x, cc5.y);
	secondPixel.y = sumPixels(aa3.x, aa4.y, aa6.x, bb3.x, bb4.y, bb6.x, cc3.x, cc4.y, cc6.x);
	secondPixel.z = sumPixels(aa3.y, aa5.x, aa6.y, bb3.y, bb5.x, bb6.y, cc3.y, cc5.x, cc6.y);

	WORD *tempSharedOut = &sharedMemoryOut[threadIdx.x * 3];

	tempSharedOut[0] = ((firstPixel.y << 8) | firstPixel.x);
	tempSharedOut[1] = ((secondPixel.x << 8) | firstPixel.z);
	tempSharedOut[2] = ((secondPixel.z << 8) | secondPixel.y);

	outputRow = &outputRow[threadIdx.x * 3];

	outputRow[0] = tempSharedOut[0];
	outputRow[1] = tempSharedOut[1];
	outputRow[2] = tempSharedOut[2];
}

__device__ WORD pack(uchar3 pixelLine)
{
	return (pixelLine.y << 8) | pixelLine.x;
}

__device__ uchar2 unpack(WORD c)
{
	uchar2 pixelLine;
	pixelLine.x = (BYTE)(c & 0xFF);
	pixelLine.y = (BYTE)((c >> 8) & 0xFF);

	return pixelLine;
}

__device__ BYTE sumPixels(BYTE a1, BYTE a2, BYTE a3, BYTE a4, BYTE a5, BYTE a6, BYTE a7, BYTE a8, BYTE a9)
{
	uint32_t result = 0;

	result = a1 + a2 + a3 + a4 + a5 + a6 + a7 + a8 + a9;

	result /= 9;

	if (result > 255) {
		result = 255;
	}

	return (BYTE)result;
}

void resizeImage(BYTE* primaryImage, BYTE* resizedImage, int imageWidth, int imageHeight) {
	for (int i = 0, int n = 0; i < imageHeight; i++, n++) {
		for (int j = 0, int m = 0; j < imageWidth; j++, m++) {
			resizedImage[(n * (imageWidth + 2) + m) * 3] = primaryImage[(i * imageWidth + j) * 3];
			resizedImage[(n * (imageWidth + 2) + m) * 3 + 1] = primaryImage[(i * imageWidth + j) * 3 + 1];
			resizedImage[(n * (imageWidth + 2) + m) * 3 + 2] = primaryImage[(i * imageWidth + j) * 3 + 2];

			if (j == 0 || j == imageWidth - 1) {
				m++;
				resizedImage[(n * (imageWidth + 2) + m) * 3] = primaryImage[(i * imageWidth + j) * 3];
				resizedImage[(n * (imageWidth + 2) + m) * 3 + 1] = primaryImage[(i * imageWidth + j) * 3 + 1];
				resizedImage[(n * (imageWidth + 2) + m) * 3 + 2] = primaryImage[(i * imageWidth + j) * 3 + 2];
			}
		}

		if (n == 0 || n == imageHeight) {
			i--;
		}
	}
}

void cudaCheckStatus(hipError_t cudaStatus) {
	if (cudaStatus != hipSuccess) {
		cout << "CUDA return error code: " << cudaStatus;
		cout << " " << hipGetErrorString(cudaStatus) << endl;
		exit(-1);
	}
}

bool checkEquality(BYTE* firstImage, BYTE* secondImage, int imageWidth, int imageHeight) {
	for (int i = 0; i < imageWidth * imageHeight; i++) {
		if (fabs(firstImage[i] - secondImage[i]) > 1) {
			return false;
		}
	}
	return true;
}