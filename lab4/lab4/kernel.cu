#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <iostream> 
#include <hip/hip_runtime.h> 
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <intrin.h>
#include <ctime>
#include <cmath>

#pragma comment(lib, "cudart") 

#define SIZE_M 123488
#define SIZE_N 1234
#define COUNT_OF_THREADS 1024
#define MAX_BLOCKS 200000

using namespace std;

void cpu_matrixOperation(short*, short*, int, int);
void cuda_matrixOperation(short*, short*, bool);
void cuda_checkStatus(hipError_t);
void fillMatrix(short*, int, int);
bool checkEquality(short*, short*, int, int);

int main() {
	auto* initMatrix = (short*)malloc(SIZE_M * SIZE_N * sizeof(short));
	auto* cpu_outMatrix = (short*)malloc(SIZE_M * SIZE_N * sizeof(short));
	auto* cuda_outMatrix = (short*)malloc(SIZE_M * SIZE_N * sizeof(short));
	auto* cuda_outMatrixSharedMemory = (short*)malloc(SIZE_M * SIZE_N * sizeof(short));

	fillMatrix(initMatrix, SIZE_M, SIZE_N);

	/*for (auto i = 0; i < SIZE_N * SIZE_M; i++) {
		printf("%2d ", initMatrix[i]);

		if ((i + 1) % SIZE_M == 0) {
			cout << endl;
		}
	}*/

	cuda_matrixOperation(initMatrix, cuda_outMatrix, false);
	cuda_matrixOperation(initMatrix, cuda_outMatrixSharedMemory, true);

	cpu_matrixOperation(initMatrix, cpu_outMatrix, SIZE_M, SIZE_N);

	if (checkEquality(cuda_outMatrix, cpu_outMatrix, SIZE_M, SIZE_N)
		&& checkEquality(cuda_outMatrixSharedMemory, cuda_outMatrix, SIZE_M, SIZE_N)) {
		cout << "Results are equals!" << endl;
	}
	else {
		cout << "Results are NOT equals!" << endl;
	}

	/*cout << endl << "Not optimize" << endl;
	for (auto i = 0; i < SIZE_N * SIZE_M; i++) {
		printf("%3d ", cuda_outMatrix[i]);
		if ((i + 1) % (SIZE_M * 2) == 0) {
			cout << endl;
		}
	}
	cout << endl << "Shared memory" << endl;
	for (auto i = 0; i < SIZE_N * SIZE_M; i++) {
		printf("%3d ", cuda_outMatrixSharedMemory[i]);
		if ((i + 1) % (SIZE_M * 2) == 0) {
			cout << endl;
		}
	}
	cout << endl << "CPU" << endl;
	for (auto i = 0; i < SIZE_N * SIZE_M; i++) {
		printf("%3d ", cpu_outMatrix[i]);
		if ((i + 1) % (SIZE_M * 2) == 0) {
			cout << endl;
		}
	}*/

	free(initMatrix);
	free(cpu_outMatrix);
	free(cuda_outMatrix);
	free(cuda_outMatrixSharedMemory);
}

__global__ void cuda_matrixOperationKernel(int* inMatrix, short* outMatrix, int numOfBlocksInRow) {
	int remainderElements = SIZE_M % COUNT_OF_THREADS;

	if (remainderElements != 0 && (blockIdx.x + 1) % numOfBlocksInRow == 0 && threadIdx.x >= remainderElements) {
		return;
	}

	int *startOfResultRow = &inMatrix[SIZE_M * (blockIdx.x / numOfBlocksInRow)];
	outMatrix = &outMatrix[SIZE_M * (blockIdx.x / numOfBlocksInRow) * 2];

	int elements = 0;
	int countOfThreads = 0;

	if (remainderElements != 0 && (blockIdx.x + 1) % numOfBlocksInRow == 0) {
		countOfThreads = remainderElements;
	}
	else {
		countOfThreads = COUNT_OF_THREADS;
	}

	if (threadIdx.x < (countOfThreads / 2)) {
		elements = startOfResultRow[(blockIdx.x % numOfBlocksInRow) * COUNT_OF_THREADS / 2 + threadIdx.x];
	}
	else {
		elements = startOfResultRow[threadIdx.x % (countOfThreads / 2) + SIZE_M / 2 + (blockIdx.x % numOfBlocksInRow) * COUNT_OF_THREADS / 2];
	}

	short firstElement = (short)elements;
	short secondElement = (short)(elements >> 16);

	int offset = COUNT_OF_THREADS * 2 * (blockIdx.x % numOfBlocksInRow);

	if (threadIdx.x < (countOfThreads / 2)) {
		outMatrix[threadIdx.x * 2 * 2 + offset] = firstElement;
		outMatrix[(threadIdx.x * 2 + 1) * 2 + offset] = secondElement;
	}
	else {
		outMatrix[(threadIdx.x - countOfThreads / 2) * 2 * 2 + 1 + offset] = firstElement;
		outMatrix[((threadIdx.x - countOfThreads / 2) * 2 + 1) * 2 + 1 + offset] = secondElement;
	}
}

__global__ void cuda_matrixSharedMemoryOperationKernel(int* inMatrix, int* outMatrix, int numOfBlocksInRow) {
	int remainderElements = SIZE_M % COUNT_OF_THREADS;

	__shared__ int sharedMemory[COUNT_OF_THREADS];
	__shared__ short sharedMemoryOut[COUNT_OF_THREADS * 2];

	if (remainderElements != 0 && (blockIdx.x + 1) % numOfBlocksInRow == 0 && threadIdx.x >= remainderElements) {
		return;
	}

	int *startOfResultRow = &inMatrix[SIZE_M * (blockIdx.x / numOfBlocksInRow)];
	outMatrix = &outMatrix[SIZE_M * (blockIdx.x / numOfBlocksInRow)];

	int countOfThreads = 0;

	if (remainderElements != 0 && (blockIdx.x + 1) % numOfBlocksInRow == 0) {
		countOfThreads = remainderElements;
	}
	else {
		countOfThreads = COUNT_OF_THREADS;
	}

	if (threadIdx.x < (countOfThreads / 2)) {
		sharedMemory[threadIdx.x] = startOfResultRow[(blockIdx.x % numOfBlocksInRow) * COUNT_OF_THREADS / 2 + threadIdx.x];
	}
	else {
		sharedMemory[threadIdx.x] = startOfResultRow[threadIdx.x % (countOfThreads / 2) + SIZE_M / 2 + (blockIdx.x % numOfBlocksInRow) * COUNT_OF_THREADS / 2];
	}

	int elements = sharedMemory[threadIdx.x];
	short firstElement = (short)elements;
	short secondElement = (short)(elements >> 16);

	int offset = COUNT_OF_THREADS * 2 * (blockIdx.x % numOfBlocksInRow);

	if (threadIdx.x < (countOfThreads / 2)) {
		sharedMemoryOut[threadIdx.x * 2 * 2] = firstElement;
		sharedMemoryOut[(threadIdx.x * 2 + 1) * 2] = secondElement;
	}
	else {
		sharedMemoryOut[(threadIdx.x - countOfThreads / 2) * 2 * 2 + 1] = firstElement;
		sharedMemoryOut[((threadIdx.x - countOfThreads / 2) * 2 + 1) * 2 + 1] = secondElement;
	}

	__syncthreads();

	outMatrix[offset / 2 + threadIdx.x] = ((int*)sharedMemoryOut)[threadIdx.x];
}

void cuda_matrixOperation(short* inMatrix, short* outMatrix, bool optimizationFlag) {
	float resultTime;

	short* device_inMatrix;
	short* device_outMatrix;

	hipEvent_t cuda_startTime;
	hipEvent_t cuda_endTime;

	cuda_checkStatus(hipEventCreate(&cuda_startTime));
	cuda_checkStatus(hipEventCreate(&cuda_endTime));

	int numOfBlocksInRow = (int)ceil((double)SIZE_M / COUNT_OF_THREADS);
	int blocksNeeded = (SIZE_N * numOfBlocksInRow) / 2;
	int maxBlocksPerIteration = MAX_BLOCKS - MAX_BLOCKS % numOfBlocksInRow;

	for (int i = 0, int times = 0; i < blocksNeeded; i += maxBlocksPerIteration, times++) {
		int blocksInIteration = (blocksNeeded - i) < maxBlocksPerIteration ? blocksNeeded - i : maxBlocksPerIteration;

		int numOfRows =  (blocksInIteration / numOfBlocksInRow) * 2;

		cuda_checkStatus(hipMalloc(&device_inMatrix, SIZE_M * numOfRows * sizeof(short)));
		cuda_checkStatus(hipMalloc(&device_outMatrix, SIZE_M  * numOfRows * sizeof(short)));
		cuda_checkStatus(hipMemcpy(
			device_inMatrix, 
			&inMatrix[SIZE_M * (maxBlocksPerIteration / numOfBlocksInRow) * 2 * times],
			SIZE_M * numOfRows * sizeof(short), hipMemcpyHostToDevice)
		);

		dim3 blockSize(COUNT_OF_THREADS);
		dim3 gridSize(blocksInIteration);

		cuda_checkStatus(hipEventRecord(cuda_startTime, NULL));

		if (optimizationFlag) {
			cuda_matrixSharedMemoryOperationKernel <<< gridSize, blockSize >>> ((int*)device_inMatrix, (int*)device_outMatrix, numOfBlocksInRow);
		}
		else {
			cuda_matrixOperationKernel <<< gridSize, blockSize >>> ((int*)device_inMatrix, device_outMatrix, numOfBlocksInRow);
		}

		cuda_checkStatus(hipPeekAtLastError());
		cuda_checkStatus(hipEventRecord(cuda_endTime, NULL));
		cuda_checkStatus(hipEventSynchronize(cuda_endTime));

		cuda_checkStatus(hipEventElapsedTime(&resultTime, cuda_startTime, cuda_endTime));

		if (optimizationFlag) {
			printf("%d: CUDA time with optimization: %lf seconds\n", times, (double)resultTime / CLOCKS_PER_SEC);
		}
		else {
			printf("%d: CUDA time: %lf seconds\n", times, (double)resultTime / CLOCKS_PER_SEC);
		}

		cuda_checkStatus(hipMemcpy(
			&outMatrix[SIZE_M * (maxBlocksPerIteration / numOfBlocksInRow) * 2 * times],
			device_outMatrix,
			SIZE_M * numOfRows * sizeof(short), hipMemcpyDeviceToHost)
		);

		cuda_checkStatus(hipFree(device_inMatrix));
		cuda_checkStatus(hipFree(device_outMatrix));
	}
}

void cpu_matrixOperation(short* inMatrix, short* outMatrix, int sizeOfM, int sizeOfN) {
	clock_t startTime, endTime;
	startTime = clock();
	for (auto i = 0; i < sizeOfM; i++) {
		for (auto j = 0; j < sizeOfN; j++) {
			int a = (j + 1) % 2 == 0 ? 1 : 0;
			outMatrix[(j / 2) * sizeOfM * 2 + a + i * 2] = inMatrix[i + sizeOfM * j];
		}
	}
	endTime = clock();
	printf("CPU time: %lf seconds\n", (double)(endTime - startTime) / CLOCKS_PER_SEC);
}

void fillMatrix(short* matrix, int sizeOfM, int sizeOfN) {
	for (int i = 0; i < sizeOfN; i++) {
		for (int j = 0; j < sizeOfM; j++) {
			matrix[sizeOfM * i + j] = rand() % 20 + 1;
		}
	}
}

void cuda_checkStatus(hipError_t cudaStatus) {
	if (cudaStatus != hipSuccess) {
		cout << "CUDA return error code: " << cudaStatus;
		cout << " " << hipGetErrorString(cudaStatus) << endl;
		exit(-1);
	}
}

bool checkEquality(short* inMatrix, short* outMatrix, int sizeOfM, int sizeOfN) {
	for (int i = 0; i < sizeOfN * sizeOfM; i++) {
		if (inMatrix[i] != outMatrix[i]) {
			return false;
		}
	}
	return true;
}