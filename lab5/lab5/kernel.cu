#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "helper_image.h"

#include <iostream> 
#include <hip/hip_runtime.h> 
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <intrin.h>
#include <windows.h>
#include <ctime>
#include <cmath>
#include <cstdlib>

#pragma comment(lib, "cudart") 

#define BLOCK_SIZE_X 1024
//#define IMAGE_WIDTH 17374
//#define IMAGE_HEIGHT 27472
#define IMAGE_WIDTH 6000
#define IMAGE_HEIGHT 4000
#define MAX_BLOCKS 200000

using namespace std;

void cpu_filterImage(BYTE*, BYTE*, int, int);
void cuda_filterImage(BYTE*, BYTE*, bool);
void cudaCheckStatus(hipError_t);
void resizeImage(BYTE*, BYTE*, int, int);
bool checkEquality(BYTE*, BYTE*, int, int);
__global__ void cuda_filterImage(BYTE*, BYTE*, size_t, size_t);
__global__ void cuda_filterImageShared(BYTE*, BYTE*, size_t, size_t);
__device__ BYTE sumPixels(BYTE, BYTE, BYTE, BYTE, BYTE, BYTE, BYTE, BYTE, BYTE);
__device__ WORD pack(uchar2);
__device__ uchar2 unpack(WORD);

int main() {
	unsigned int imageWidth = 0, imageHeight = 0, channels = 0;

	const char primaryImagePath[] = "D:\\image.pgm";
	const char outputImageCpuPath[] = "D:\\imageCPU.pgm";
	const char outputImageGpuPath[] = "D:\\imageGPU.pgm";
	const char outputImageGpuSharedPath[] = "D:\\imageGPUshared.pgm";

	BYTE *primaryImage = NULL;

	__loadPPM(primaryImagePath, &primaryImage, &imageWidth, &imageHeight, &channels);

	auto* outputImageCpu = (BYTE*)malloc(imageWidth * imageHeight * sizeof(BYTE));
	auto* outputImageGpu = (BYTE*)malloc(imageWidth * imageHeight * sizeof(BYTE));
	auto* outputImageGpuShared = (BYTE*)malloc(imageWidth * imageHeight * sizeof(BYTE));
	auto* resizedImage = (BYTE*)malloc((imageWidth + 2) * (imageHeight + 2) * sizeof(BYTE));

	resizeImage(primaryImage, resizedImage, imageWidth, imageHeight);

	cuda_filterImage(resizedImage, outputImageGpuShared, true);
	cuda_filterImage(resizedImage, outputImageGpu, false);
	cpu_filterImage(resizedImage, outputImageCpu, imageWidth, imageHeight);

	__savePPM(outputImageCpuPath, outputImageCpu, imageWidth, imageHeight, channels);
	__savePPM(outputImageGpuPath, outputImageGpu, imageWidth, imageHeight, channels);
	__savePPM(outputImageGpuSharedPath, outputImageGpuShared, imageWidth, imageHeight, channels);

	cout << "Start compare" << endl;

	if (checkEquality(outputImageCpu, outputImageGpu, IMAGE_WIDTH, IMAGE_HEIGHT)
		&& checkEquality(outputImageGpu, outputImageGpuShared, IMAGE_WIDTH, IMAGE_HEIGHT)) {
		cout << "Results are equals!" << endl;
	}
	else {
		cout << "Results are NOT equals!" << endl;
	}

	free(primaryImage);
	free(resizedImage);
	free(outputImageCpu);
	free(outputImageGpu);
	free(outputImageGpuShared);
}

void cuda_filterImage(BYTE* inMatrix, BYTE* outMatrix, bool optimizationFlag) {
	float resultTime;

	BYTE* device_inMatrix;
	BYTE* device_outMatrix;

	hipEvent_t cuda_startTime;
	hipEvent_t cuda_endTime;

	cudaCheckStatus(hipEventCreate(&cuda_startTime));
	cudaCheckStatus(hipEventCreate(&cuda_endTime));

	int numOfBlocksInRow = (int)ceil((double)(IMAGE_WIDTH) / (BLOCK_SIZE_X * 2));
	int numOfBlockInColumn = IMAGE_HEIGHT;
	int blocksNeeded = numOfBlockInColumn * numOfBlocksInRow;
	int maxBlocksPerIteration = MAX_BLOCKS - MAX_BLOCKS % numOfBlocksInRow;

	for (int i = 0, int times = 1; i < blocksNeeded; i += maxBlocksPerIteration, times++) {
		int blocksInIteration = (blocksNeeded - i) < maxBlocksPerIteration ? blocksNeeded - i : maxBlocksPerIteration;
		size_t pitchInMatrix = 0, pitchOutMatrix = 0;
		int gridSizeY = blocksInIteration / numOfBlocksInRow;
		int gridSizeX = numOfBlocksInRow;

		cudaCheckStatus(hipMallocPitch((void**)&device_inMatrix, &pitchInMatrix, IMAGE_WIDTH + 2, gridSizeY + 2));
		cudaCheckStatus(hipMallocPitch((void**)&device_outMatrix, &pitchOutMatrix, IMAGE_WIDTH, gridSizeY));
		cudaCheckStatus(hipMemcpy2D(
			device_inMatrix, pitchInMatrix,
			inMatrix, IMAGE_WIDTH + 2,
			IMAGE_WIDTH + 2, gridSizeY + 2,
			hipMemcpyHostToDevice));

		dim3 blockSize(BLOCK_SIZE_X);
		dim3 gridSize(gridSizeX, gridSizeY);

		cudaCheckStatus(hipEventRecord(cuda_startTime, NULL));

		if (optimizationFlag) {
			cuda_filterImageShared << < gridSize, blockSize >> > (device_inMatrix, device_outMatrix, pitchInMatrix, pitchOutMatrix);
		}
		else {
			cuda_filterImage << < gridSize, blockSize >> > (device_inMatrix, device_outMatrix, pitchInMatrix, pitchOutMatrix);
		}

		cudaCheckStatus(hipPeekAtLastError());
		cudaCheckStatus(hipEventRecord(cuda_endTime, NULL));
		cudaCheckStatus(hipEventSynchronize(cuda_endTime));

		cudaCheckStatus(hipEventElapsedTime(&resultTime, cuda_startTime, cuda_endTime));

		if (optimizationFlag) {
			printf("%d: CUDA time with optimization: %lf seconds\n", times, (double)resultTime / CLOCKS_PER_SEC);
		}
		else {
			printf("%d: CUDA time: %lf seconds\n", times, (double)resultTime / CLOCKS_PER_SEC);
		}

		cudaCheckStatus(hipMemcpy2D(
			outMatrix, IMAGE_WIDTH,
			device_outMatrix, pitchOutMatrix,
			IMAGE_WIDTH, gridSizeY,
			hipMemcpyDeviceToHost)
		);

		inMatrix = &inMatrix[(IMAGE_WIDTH + 2) * gridSizeY * times];
		outMatrix = &outMatrix[IMAGE_WIDTH * gridSizeY * times];

		cudaCheckStatus(hipFree(device_inMatrix));
		cudaCheckStatus(hipFree(device_outMatrix));
	}
}

void cpu_filterImage(BYTE* primaryImage, BYTE* outputImage, int imageWidth, int imageHeight) {
	primaryImage = &primaryImage[imageWidth + 2 + 1];

	clock_t startTime, endTime;
	startTime = clock();
	for (auto i = 0; i < imageHeight; i++) {
		for (auto j = 0; j < imageWidth; j++) {
			short sum = 0;

			sum += primaryImage[i * (imageWidth + 2) + j];
			sum += primaryImage[i * (imageWidth + 2) + j + 1];
			sum += primaryImage[i * (imageWidth + 2) + j - 1];

			sum += primaryImage[(i + 1) * (imageWidth + 2) + j];
			sum += primaryImage[(i + 1) * (imageWidth + 2) + j + 1];
			sum += primaryImage[(i + 1) * (imageWidth + 2) + j - 1];

			sum += primaryImage[(i - 1) * (imageWidth + 2) + j];
			sum += primaryImage[(i - 1) * (imageWidth + 2) + j + 1];
			sum += primaryImage[(i - 1) * (imageWidth + 2) + j - 1];

			sum = sum / 9;

			if (sum > 255) {
				sum = 255;
			}
			else if (sum < 0) {
				sum = 0;
			}

			outputImage[i * imageWidth + j] = (unsigned char)sum;
		}
	}
	endTime = clock();
	printf("-  CPU time: %lf seconds\n", (double)(endTime - startTime) / CLOCKS_PER_SEC);
}

__global__ void cuda_filterImage(BYTE* inMatrix, BYTE* outMatrix, size_t pitchInMatrix, size_t pitchOutMatrix) {
	int remainderElements = (IMAGE_WIDTH % (blockDim.x * 2)) / 2;

	if (remainderElements != 0 && (blockIdx.x + 1) % gridDim.x == 0 && threadIdx.x >= remainderElements) {
		return;
	}

	WORD *startOfProcessingRow = (WORD*)&inMatrix[pitchInMatrix * blockIdx.y + blockIdx.x * blockDim.x * 2 + threadIdx.x * 2];

	WORD a1 = startOfProcessingRow[0];
	WORD a2 = startOfProcessingRow[1];
	WORD b1 = startOfProcessingRow[pitchInMatrix / 2];
	WORD b2 = startOfProcessingRow[pitchInMatrix / 2 + 1];
	WORD c1 = startOfProcessingRow[pitchInMatrix];
	WORD c2 = startOfProcessingRow[pitchInMatrix + 1];

	uchar2 aa1 = unpack(a1);
	uchar2 aa2 = unpack(a2);
	uchar2 bb1 = unpack(b1);
	uchar2 bb2 = unpack(b2);
	uchar2 cc1 = unpack(c1);
	uchar2 cc2 = unpack(c2);

	BYTE firstPixel = sumPixels(aa1.x, aa1.y, aa2.x, bb1.x, bb1.y, bb2.x, cc1.x, cc1.y, cc2.x);
	BYTE secondPixel = sumPixels(aa1.y, aa2.x, aa2.y, bb1.y, bb2.x, bb2.y, cc1.y, cc2.x, cc2.y);

	outMatrix[blockIdx.y * pitchOutMatrix + blockIdx.x * blockDim.x * 2 + threadIdx.x * 2] = firstPixel;
	outMatrix[blockIdx.y * pitchOutMatrix + blockIdx.x * blockDim.x * 2 + threadIdx.x * 2 + 1] = secondPixel;
}

__global__ void cuda_filterImageShared(BYTE* inMatrix, BYTE* outMatrix, size_t pitchInMatrix, size_t pitchOutMatrix) {
	int remainderElements = (IMAGE_WIDTH % (blockDim.x * 2)) / 2;

	if (remainderElements != 0 && (blockIdx.x + 1) % gridDim.x == 0 && threadIdx.x >= remainderElements) {
		return;
	}

	__shared__ WORD sharedMemoryIn[3][BLOCK_SIZE_X + 1];
	__shared__ WORD sharedMemoryOut[BLOCK_SIZE_X];

	WORD *startOfProcessingRow = (WORD*)&inMatrix[blockIdx.y * pitchInMatrix + blockIdx.x * blockDim.x * 2 + threadIdx.x * 2];
	WORD *outputRow = (WORD*)&outMatrix[blockIdx.y * pitchOutMatrix + blockIdx.x * blockDim.x * 2];

	if (threadIdx.x == 0) {
		sharedMemoryIn[0][threadIdx.x] = startOfProcessingRow[0];
		sharedMemoryIn[1][threadIdx.x] = startOfProcessingRow[pitchInMatrix / 2];
		sharedMemoryIn[2][threadIdx.x] = startOfProcessingRow[pitchInMatrix];
	}

	sharedMemoryIn[0][threadIdx.x + 1] = startOfProcessingRow[1];
	sharedMemoryIn[1][threadIdx.x + 1] = startOfProcessingRow[pitchInMatrix / 2 + 1];
	sharedMemoryIn[2][threadIdx.x + 1] = startOfProcessingRow[pitchInMatrix + 1];

	__syncthreads();

	uchar2 aa1 = unpack(sharedMemoryIn[0][threadIdx.x]);
	uchar2 aa2 = unpack(sharedMemoryIn[0][threadIdx.x + 1]);

	uchar2 bb1 = unpack(sharedMemoryIn[1][threadIdx.x]);
	uchar2 bb2 = unpack(sharedMemoryIn[1][threadIdx.x + 1]);

	uchar2 cc1 = unpack(sharedMemoryIn[2][threadIdx.x]);
	uchar2 cc2 = unpack(sharedMemoryIn[2][threadIdx.x + 1]);

	uchar2 pixels;
	pixels.x = sumPixels(aa1.x, aa1.y, aa2.x, bb1.x, bb1.y, bb2.x, cc1.x, cc1.y, cc2.x);
	pixels.y = sumPixels(aa1.y, aa2.x, aa2.y, bb1.y, bb2.x, bb2.y, cc1.y, cc2.x, cc2.y);

	sharedMemoryOut[threadIdx.x] = pack(pixels);

	outputRow[threadIdx.x] = sharedMemoryOut[threadIdx.x];
}

__device__ WORD pack(uchar2 pixels)
{
	return (pixels.y << 8) | pixels.x;
}

__device__ uchar2 unpack(WORD c)
{
	uchar2 pixelLine;
	pixelLine.x = (BYTE)(c & 0xFF);
	pixelLine.y = (BYTE)((c >> 8) & 0xFF);

	return pixelLine;
}

__device__ BYTE sumPixels(BYTE a1, BYTE a2, BYTE a3, BYTE a4, BYTE a5, BYTE a6, BYTE a7, BYTE a8, BYTE a9)
{
	uint32_t result = 0;

	result = a1 + a2 + a3 + a4 + a5 + a6 + a7 + a8 + a9;

	result /= 9;

	if (result > 255) {
		result = 255;
	}

	return (BYTE)result;
}

void resizeImage(BYTE* primaryImage, BYTE* resizedImage, int imageWidth, int imageHeight) {
	for (int i = 0, int n = 0; i < imageHeight; i++, n++) {
		for (int j = 0, int m = 0; j < imageWidth; j++, m++) {
			resizedImage[n * (imageWidth + 2) + m] = primaryImage[i * imageWidth + j];

			if (j == 0 || j == imageWidth - 1) {
				m++;
				resizedImage[n * (imageWidth + 2) + m] = primaryImage[i * imageWidth + j];
			}
		}

		if (n == 0 || n == imageHeight) {
			i--;
		}
	}
}

void cudaCheckStatus(hipError_t cudaStatus) {
	if (cudaStatus != hipSuccess) {
		cout << "CUDA return error code: " << cudaStatus;
		cout << " " << hipGetErrorString(cudaStatus) << endl;
		exit(-1);
	}
}

bool checkEquality(BYTE* firstImage, BYTE* secondImage, int imageWidth, int imageHeight) {
	for (int i = 0; i < imageWidth * imageHeight; i++) {
		if (fabs(firstImage[i] - secondImage[i]) > 1) {
			return false;
		}
	}
	return true;
}
